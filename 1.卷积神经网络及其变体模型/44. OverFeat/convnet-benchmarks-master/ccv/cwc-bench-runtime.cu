#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

extern "C" void cwc_bench_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int batch = params.mini_batch;
	int i;
	const int device_id = 0;
	_cwc_convnet_alloc_reserved_both(convnet, batch, 0, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	for (i = 0; i < convnet->rows * convnet->cols * convnet->channels; i++)
		convnet->mean_activity->data.f32[i] = 128;
	cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, 0, ccv_size(128, 128), 128, 128, convnet->rows, convnet->cols, convnet->channels, 1000, 0, batch, 0, batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpy(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * batch, hipMemcpyHostToDevice);

	hipEvent_t overallStart;
	hipEvent_t overallStop;
	hipEventCreate(&overallStart);
	hipEventCreate(&overallStop);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.forward.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.forward.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.forward.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 1)->vary.convolutional.forward.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 1)->vary.convolutional.forward.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 1)->vary.convolutional.forward.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 2)->vary.convolutional.forward.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 2)->vary.convolutional.forward.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 2)->vary.convolutional.forward.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.forward.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.forward.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.forward.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 4)->vary.convolutional.forward.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 4)->vary.convolutional.forward.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 4)->vary.convolutional.forward.z = 32;
	hipEventRecord(overallStart, context->device[0].data_stream);
	for (i = 0; i < convnet->count; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[0].layers + i;
		hipEventRecord(start, context->device[0].data_stream);
		_cwc_convnet_layer_forward_propagate(layer, 0, i, layer->input.matrix.rows, layer->input.matrix.cols, batch, 0, i == 0 ? context->device[0].input : GPU(convnet)->device[0].forwards[i - 1], GPU(convnet)->device[0].forwards[i], GPU(convnet)->device[0].denoms[i], GPU(convnet)->device[0].unit, context);
		hipEventRecord(stop, context->device[0].data_stream);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		if (layer->type == CCV_CONVNET_CONVOLUTIONAL)
			printf("%d %d %d, elapsed time for layer %d fprop: %f milliseconds\n", EXTRA(layer)->vary.convolutional.forward.x, EXTRA(layer)->vary.convolutional.forward.y, EXTRA(layer)->vary.convolutional.forward.z, i + 1, elapsed_time);
		else
			printf("elapsed time for layer %d fprop: %f milliseconds\n", i + 1, elapsed_time);
	}
	hipEventRecord(overallStop, context->device[0].data_stream);
	hipEventSynchronize(overallStop);
	hipEventElapsedTime(&elapsed_time, overallStart, overallStop);
	printf("forward pass %f milliseconds\n", elapsed_time);

	/*
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.backward.coefficient.x = 1;
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.backward.coefficient.y = 3;
	EXTRA(GPU(convnet)->device[0].layers + 0)->vary.convolutional.backward.coefficient.z = 1;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.coefficient.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.coefficient.y = 4;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.coefficient.z = 16;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.gradient.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.gradient.y = 6;
	EXTRA(GPU(convnet)->device[0].layers + 3)->vary.convolutional.backward.gradient.z = 24;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.coefficient.x = 8;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.coefficient.y = 3;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.coefficient.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.gradient.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.gradient.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 6)->vary.convolutional.backward.gradient.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.coefficient.x = 8;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.coefficient.y = 3;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.coefficient.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.gradient.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.gradient.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 7)->vary.convolutional.backward.gradient.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.coefficient.x = 8;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.coefficient.y = 4;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.coefficient.z = 32;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.gradient.x = 4;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.gradient.y = 8;
	EXTRA(GPU(convnet)->device[0].layers + 8)->vary.convolutional.backward.gradient.z = 32;
	float* a = 0;
	hipMalloc(&a, sizeof(float) * 1000 * batch);
	hipMemcpy(a, GPU(convnet)->device[0].forwards[convnet->count - 1], sizeof(float) * 1000 * batch, hipMemcpyDeviceToDevice);
	hipEventRecord(overallStart, context->device[0].data_stream);
	for (i = convnet->count - 1; i >= 0; i--)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[0].layers + i;
		ccv_convnet_layer_t* configuration = GPU(convnet)->device[0].configurations + i;
		hipEventRecord(start, context->device[0].data_stream);
		switch (layer->type)
		{
			case CCV_CONVNET_CONVOLUTIONAL:
				if (context->device[0].dor[i])
				{
					int out_rows, out_cols, out_partition;
					ccv_convnet_make_output(layer, layer->input.matrix.rows, layer->input.matrix.cols, &out_rows, &out_cols, &out_partition);
					_cwc_kern_mute_neuron
					<<<out_rows * out_cols * layer->net.convolutional.count, batch, 0, context->device[0].data_stream>>>
					(i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], context->device[0].dor[i]);
				}
				_cwc_convnet_convolutional_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], GPU(convnet)->device[0].forwards[i], i > 0 ? GPU(convnet)->device[0].forwards[i - 1] : context->device[0].input, GPU(convnet)->device[0].backwards[i], configuration, GPU(convnet)->device[0].scratch, GPU(convnet)->device[0].unit, context->device[0].data_stream, context->device[0].data_cublas);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_FULL_CONNECT:
				if (context->device[0].dor[i])
					_cwc_kern_mute_neuron
					<<<layer->net.full_connect.count, batch, 0, context->device[0].data_stream>>>
					(i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], context->device[0].dor[i]);
				_cwc_convnet_full_connect_backward_propagate(layer, batch,  i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], GPU(convnet)->device[0].forwards[i], i > 0 ? GPU(convnet)->device[0].forwards[i - 1] : context->device[0].input, GPU(convnet)->device[0].backwards[i], GPU(convnet)->device[0].unit, configuration, context->device[0].data_stream, context->device[0].data_cublas);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_LOCAL_RESPONSE_NORM:
				_cwc_convnet_rnorm_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], GPU(convnet)->device[0].forwards[i], i > 0 ? GPU(convnet)->device[0].forwards[i - 1] : context->device[0].input, GPU(convnet)->device[0].denoms[i], GPU(convnet)->device[0].backwards[i], context->device[0].data_stream);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_MAX_POOL:
				_cwc_convnet_max_pool_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], GPU(convnet)->device[0].forwards[i], i > 0 ? GPU(convnet)->device[0].forwards[i - 1] : context->device[0].input, GPU(convnet)->device[0].backwards[i], context->device[0].data_stream);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_AVERAGE_POOL:
				_cwc_convnet_average_pool_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->device[0].backwards[i + 1], GPU(convnet)->device[0].backwards[i], context->device[0].data_stream);
				assert(hipGetLastError() == hipSuccess);
				break;
		}
		hipEventRecord(stop, context->device[0].data_stream);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		if (layer->type == CCV_CONVNET_CONVOLUTIONAL)
			printf("%d %d %d, %d %d %d, elapsed time for layer %d bprop: %f milliseconds\n", EXTRA(layer)->vary.convolutional.backward.coefficient.x, EXTRA(layer)->vary.convolutional.backward.coefficient.y, EXTRA(layer)->vary.convolutional.backward.coefficient.z, EXTRA(layer)->vary.convolutional.backward.gradient.x, EXTRA(layer)->vary.convolutional.backward.gradient.y, EXTRA(layer)->vary.convolutional.backward.gradient.z, i + 1, elapsed_time);
		else
			printf("elapsed time for layer %d bprop: %f milliseconds\n", i + 1, elapsed_time);
	}
	hipEventRecord(overallStop, context->device[0].data_stream);
	hipEventSynchronize(overallStop);
	hipEventElapsedTime(&elapsed_time, overallStart, overallStop);
	printf("backward pass %f milliseconds\n", elapsed_time);
	hipFree(a);
	*/
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(overallStart);
	hipEventDestroy(overallStop);
}
