/**
 *  Copyright (c) 2019-2022 by Contributors
 * @file array/cuda/uvm/array_index_select_uvm.cu
 * @brief Array index select GPU implementation
 */
#include <dgl/array.h>

#include "../../../runtime/cuda/cuda_common.h"
#include "../array_index_select.cuh"
#include "../utils.h"
#include "./array_index_select_uvm.cuh"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename DType, typename IdType>
NDArray IndexSelectCPUFromGPU(NDArray array, IdArray index) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  int64_t num_feat = 1;
  std::vector<int64_t> shape{len};

  CHECK(array.IsPinned());
  const DType* array_data = static_cast<DType*>(cuda::GetDevicePointer(array));
  CHECK_EQ(index->ctx.device_type, kDGLCUDA);

  for (int d = 1; d < array->ndim; ++d) {
    num_feat *= array->shape[d];
    shape.emplace_back(array->shape[d]);
  }

  NDArray ret = NDArray::Empty(shape, array->dtype, index->ctx);
  if (len == 0 || arr_len * num_feat == 0) return ret;
  DType* ret_data = static_cast<DType*>(ret->data);

  auto res = Sort(index, cuda::_NumberOfBits(arr_len));
  const IdType* idx_data = static_cast<IdType*>(res.first->data);
  const int64_t* perm_data = static_cast<int64_t*>(res.second->data);

  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(
        IndexSelectSingleKernel, nb, nt, 0, stream, array_data, idx_data, len,
        arr_len, ret_data, perm_data);
  } else {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    if (num_feat * sizeof(DType) < 2 * CACHE_LINE_SIZE) {
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernel, grid, block, 0, stream, array_data, num_feat,
          idx_data, len, arr_len, ret_data, perm_data);
    } else {
      CUDA_KERNEL_CALL(
          IndexSelectMultiKernelAligned, grid, block, 0, stream, array_data,
          num_feat, idx_data, len, arr_len, ret_data, perm_data);
    }
  }
  return ret;
}

// floating point types are treated as their equal width integer types
template NDArray IndexSelectCPUFromGPU<int8_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int8_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int16_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int16_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int32_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int32_t, int64_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int64_t, int32_t>(NDArray, IdArray);
template NDArray IndexSelectCPUFromGPU<int64_t, int64_t>(NDArray, IdArray);

template <typename DType, typename IdType>
void IndexScatterGPUToCPU(NDArray dest, IdArray index, NDArray source) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const DType* source_data = static_cast<DType*>(source->data);
  const IdType* idx_data = static_cast<IdType*>(index->data);
  const int64_t arr_len = dest->shape[0];
  const int64_t len = index->shape[0];
  int64_t num_feat = 1;
  std::vector<int64_t> shape{len};

  CHECK(dest.IsPinned());
  DType* dest_data = static_cast<DType*>(cuda::GetDevicePointer(dest));
  CHECK_EQ(index->ctx.device_type, kDGLCUDA);
  CHECK_EQ(source->ctx.device_type, kDGLCUDA);

  for (int d = 1; d < source->ndim; ++d) {
    num_feat *= source->shape[d];
  }

  if (len == 0) return;

  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(
        IndexScatterSingleKernel, nb, nt, 0, stream, source_data, idx_data, len,
        arr_len, dest_data);
  } else {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    CUDA_KERNEL_CALL(
        IndexScatterMultiKernel, grid, block, 0, stream, source_data, num_feat,
        idx_data, len, arr_len, dest_data);
  }
}

// floating point types are treated as their equal width integer types
template void IndexScatterGPUToCPU<int8_t, int32_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int8_t, int64_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int16_t, int32_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int16_t, int64_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int32_t, int32_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int32_t, int64_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int64_t, int32_t>(NDArray, IdArray, NDArray);
template void IndexScatterGPUToCPU<int64_t, int64_t>(NDArray, IdArray, NDArray);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
