#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2022 by Contributors
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 * \file gpu_cache.cu
 * \brief Implementation of wrapper HugeCTR gpu_cache routines.
 */

#ifndef DGL_RUNTIME_CUDA_GPU_CACHE_H_
#define DGL_RUNTIME_CUDA_GPU_CACHE_H_

#include <hip/hip_runtime.h>
#include <dgl/array.h>
#include <dgl/aten/array_ops.h>
#include <dgl/packed_func_ext.h>
#include <dgl/runtime/container.h>
#include <dgl/runtime/device_api.h>
#include <dgl/runtime/object.h>
#include <dgl/runtime/registry.h>

#include <nv_gpu_cache.hpp>

#include "../../runtime/cuda/cuda_common.h"

namespace dgl {
namespace runtime {
namespace cuda {

template <typename key_t>
class GpuCache : public runtime::Object {
  constexpr static int set_associativity = 2;
  constexpr static int WARP_SIZE = 32;
  constexpr static int bucket_size = WARP_SIZE * set_associativity;
  using gpu_cache_t = gpu_cache::gpu_cache<
      key_t, uint64_t, std::numeric_limits<key_t>::max(), set_associativity,
      WARP_SIZE>;

 public:
  static constexpr const char *_type_key =
      sizeof(key_t) == 4 ? "cuda.GpuCache32" : "cuda.GpuCache64";
  DGL_DECLARE_OBJECT_TYPE_INFO(GpuCache, Object);

  GpuCache(size_t num_items, size_t num_feats)
      : num_feats(num_feats),
        cache(std::make_unique<gpu_cache_t>(
            (num_items + bucket_size - 1) / bucket_size, num_feats)) {
    CUDA_CALL(hipGetDevice(&cuda_device));
  }

  std::tuple<NDArray, IdArray, IdArray> Query(IdArray keys) {
    const auto &ctx = keys->ctx;
    hipStream_t stream = dgl::runtime::getCurrentCUDAStream();
    auto device = dgl::runtime::DeviceAPI::Get(ctx);
    CHECK_EQ(ctx.device_type, kDGLCUDA)
        << "The keys should be on a CUDA device";
    CHECK_EQ(ctx.device_id, cuda_device)
        << "The keys should be on the correct CUDA device";
    CHECK_EQ(keys->ndim, 1)
        << "The tensor of requested indices must be of dimension one.";
    NDArray values = NDArray::Empty(
        {keys->shape[0], (int64_t)num_feats}, DGLDataType{kDGLFloat, 32, 1},
        ctx);
    IdArray missing_index = aten::NewIdArray(keys->shape[0], ctx, 64);
    IdArray missing_keys =
        aten::NewIdArray(keys->shape[0], ctx, sizeof(key_t) * 8);
    size_t *missing_len =
        static_cast<size_t *>(device->AllocWorkspace(ctx, sizeof(size_t)));
    cache->Query(
        static_cast<const key_t *>(keys->data), keys->shape[0],
        static_cast<float *>(values->data),
        static_cast<uint64_t *>(missing_index->data),
        static_cast<key_t *>(missing_keys->data), missing_len, stream);
    size_t missing_len_host;
    device->CopyDataFromTo(
        missing_len, 0, &missing_len_host, 0, sizeof(missing_len_host), ctx,
        DGLContext{kDGLCPU, 0}, keys->dtype);
    device->FreeWorkspace(ctx, missing_len);
    missing_index = missing_index.CreateView(
        {(int64_t)missing_len_host}, missing_index->dtype);
    missing_keys =
        missing_keys.CreateView({(int64_t)missing_len_host}, keys->dtype);
    return std::make_tuple(values, missing_index, missing_keys);
  }

  void Replace(IdArray keys, NDArray values) {
    hipStream_t stream = dgl::runtime::getCurrentCUDAStream();
    CHECK_EQ(keys->ctx.device_type, kDGLCUDA)
        << "The keys should be on a CUDA device";
    CHECK_EQ(keys->ctx.device_id, cuda_device)
        << "The keys should be on the correct CUDA device";
    CHECK_EQ(values->ctx.device_type, kDGLCUDA)
        << "The values should be on a CUDA device";
    CHECK_EQ(values->ctx.device_id, cuda_device)
        << "The values should be on the correct CUDA device";
    CHECK_EQ(keys->shape[0], values->shape[0])
        << "First dimensions of keys and values must match";
    CHECK_EQ(values->shape[1], num_feats) << "Embedding dimension must match";
    cache->Replace(
        static_cast<const key_t *>(keys->data), keys->shape[0],
        static_cast<const float *>(values->data), stream);
  }

 private:
  size_t num_feats;
  std::unique_ptr<gpu_cache_t> cache;
  int cuda_device;
};

static_assert(sizeof(unsigned int) == 4);
DGL_DEFINE_OBJECT_REF(GpuCacheRef32, GpuCache<unsigned int>);
// The cu file in HugeCTR gpu cache uses unsigned int and long long.
// Changing to int64_t results in a mismatch of template arguments.
static_assert(sizeof(long long) == 8);                      // NOLINT
DGL_DEFINE_OBJECT_REF(GpuCacheRef64, GpuCache<long long>);  // NOLINT

/* CAPI **********************************************************************/

using namespace dgl::runtime;

DGL_REGISTER_GLOBAL("cuda._CAPI_DGLGpuCacheCreate")
    .set_body([](DGLArgs args, DGLRetValue *rv) {
      const size_t num_items = args[0];
      const size_t num_feats = args[1];
      const int num_bits = args[2];

      if (num_bits == 32)
        *rv = GpuCacheRef32(
            std::make_shared<GpuCache<unsigned int>>(num_items, num_feats));
      else
        *rv = GpuCacheRef64(std::make_shared<GpuCache<long long>>(  // NOLINT
            num_items, num_feats));
    });

DGL_REGISTER_GLOBAL("cuda._CAPI_DGLGpuCacheQuery")
    .set_body([](DGLArgs args, DGLRetValue *rv) {
      IdArray keys = args[1];

      List<ObjectRef> ret;
      if (keys->dtype.bits == 32) {
        GpuCacheRef32 cache = args[0];
        auto result = cache->Query(keys);

        ret.push_back(Value(MakeValue(std::get<0>(result))));
        ret.push_back(Value(MakeValue(std::get<1>(result))));
        ret.push_back(Value(MakeValue(std::get<2>(result))));
      } else {
        GpuCacheRef64 cache = args[0];
        auto result = cache->Query(keys);

        ret.push_back(Value(MakeValue(std::get<0>(result))));
        ret.push_back(Value(MakeValue(std::get<1>(result))));
        ret.push_back(Value(MakeValue(std::get<2>(result))));
      }

      *rv = ret;
    });

DGL_REGISTER_GLOBAL("cuda._CAPI_DGLGpuCacheReplace")
    .set_body([](DGLArgs args, DGLRetValue *rv) {
      IdArray keys = args[1];
      NDArray values = args[2];

      if (keys->dtype.bits == 32) {
        GpuCacheRef32 cache = args[0];
        cache->Replace(keys, values);
      } else {
        GpuCacheRef64 cache = args[0];
        cache->Replace(keys, values);
      }

      *rv = List<ObjectRef>{};
    });

}  // namespace cuda
}  // namespace runtime
}  // namespace dgl

#endif
