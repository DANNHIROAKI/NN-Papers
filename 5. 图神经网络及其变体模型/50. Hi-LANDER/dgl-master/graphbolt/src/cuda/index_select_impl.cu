#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 * @file cuda/index_select_impl.cu
 * @brief Index select operator implementation on CUDA.
 */
#include <c10/cuda/CUDAException.h>
#include <torch/script.h>

#include <numeric>

#include "../index_select.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

/** @brief Index select operator implementation for feature size 1. */
template <typename DType, typename IdType>
__global__ void IndexSelectSingleKernel(
    const DType* input, const int64_t input_len, const IdType* index,
    const int64_t output_len, DType* output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  while (out_row_index < output_len) {
    assert(index[out_row_index] >= 0 && index[out_row_index] < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    output[out_row] = input[index[out_row_index]];
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernel(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t column = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (column < feature_size) {
      output[out_row * feature_size + column] =
          input[in_row * feature_size + column];
      column += blockDim.x;
    }
    out_row_index += stride;
  }
}

/**
 * @brief Index select operator implementation for feature size > 1.
 *
 * @note This is a cross-device access version of IndexSelectMultiKernel. Since
 * the memory access over PCIe is more sensitive to the data access aligment
 * (cacheline), we need a separate version here.
 */
template <typename DType, typename IdType>
__global__ void IndexSelectMultiKernelAligned(
    const DType* const input, const int64_t input_len,
    const int64_t feature_size, const IdType* const index,
    const int64_t output_len, DType* const output,
    const int64_t* permutation = nullptr) {
  int64_t out_row_index = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row_index < output_len) {
    int64_t col = threadIdx.x;
    const int64_t in_row = index[out_row_index];
    assert(in_row >= 0 && in_row < input_len);
    const int64_t idx_offset =
        ((uint64_t)(&input[in_row * feature_size]) % GPU_CACHE_LINE_SIZE) /
        sizeof(DType);
    col = col - idx_offset;
    const auto out_row =
        permutation ? permutation[out_row_index] : out_row_index;
    while (col < feature_size) {
      if (col >= 0)
        output[out_row * feature_size + col] =
            input[in_row * feature_size + col];
      col += blockDim.x;
    }
    out_row_index += stride;
  }
}

template <typename DType, typename IdType>
torch::Tensor UVAIndexSelectImpl_(torch::Tensor input, torch::Tensor index) {
  const int64_t input_len = input.size(0);
  const int64_t return_len = index.size(0);
  const int64_t feature_size = std::accumulate(
      input.sizes().begin() + 1, input.sizes().end(), 1, std::multiplies<>());
  torch::Tensor ret = torch::empty(
      {return_len, feature_size}, torch::TensorOptions()
                                      .dtype(input.dtype())
                                      .device(c10::DeviceType::CUDA));
  DType* input_ptr = input.data_ptr<DType>();
  DType* ret_ptr = ret.data_ptr<DType>();

  // Sort the index to improve the memory access pattern.
  torch::Tensor sorted_index, permutation;
  std::tie(sorted_index, permutation) = torch::sort(index);
  const IdType* index_sorted_ptr = sorted_index.data_ptr<IdType>();
  const int64_t* permutation_ptr = permutation.data_ptr<int64_t>();

  hipStream_t stream = 0;

  if (feature_size == 1) {
    // Use a single thread to process each output row to avoid wasting threads.
    const int num_threads = cuda::FindNumThreads(return_len);
    const int num_blocks = (return_len + num_threads - 1) / num_threads;
    IndexSelectSingleKernel<<<num_blocks, num_threads, 0, stream>>>(
        input_ptr, input_len, index_sorted_ptr, return_len, ret_ptr,
        permutation_ptr);
  } else {
    dim3 block(512, 1);
    while (static_cast<int64_t>(block.x) >= 2 * feature_size) {
      block.x >>= 1;
      block.y <<= 1;
    }
    const dim3 grid((return_len + block.y - 1) / block.y);
    if (feature_size * sizeof(DType) <= GPU_CACHE_LINE_SIZE) {
      // When feature size is smaller than GPU cache line size, use unaligned
      // version for less SM usage, which is more resource efficient.
      IndexSelectMultiKernel<<<grid, block, 0, stream>>>(
          input_ptr, input_len, feature_size, index_sorted_ptr, return_len,
          ret_ptr, permutation_ptr);
    } else {
      // Use aligned version to improve the memory access pattern.
      IndexSelectMultiKernelAligned<<<grid, block, 0, stream>>>(
          input_ptr, input_len, feature_size, index_sorted_ptr, return_len,
          ret_ptr, permutation_ptr);
    }
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  auto return_shape = std::vector<int64_t>({return_len});
  return_shape.insert(
      return_shape.end(), input.sizes().begin() + 1, input.sizes().end());
  ret = ret.reshape(return_shape);
  return ret;
}

/**
 * @brief UVA index select operator implementation on CUDA.
 *
 * The supporting input types are: float, double, int, int64_t.
 * The supporting index types are: int, int64_t.
 */
torch::Tensor UVAIndexSelectImpl(torch::Tensor input, torch::Tensor index) {
  return AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Int, at::ScalarType::Long, input.scalar_type(),
      "UVAIndexSelectImpl", [&] {
        return AT_DISPATCH_INDEX_TYPES(
            index.scalar_type(), "UVAIndexSelectImpl", [&] {
              return UVAIndexSelectImpl_<scalar_t, index_t>(input, index);
            });
      });
}

}  //  namespace ops
}  //  namespace graphbolt
