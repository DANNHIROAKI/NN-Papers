#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2021 by contributors.
 * @file array/cuda/spmat_op_impl_coo.cu
 * @brief COO operator GPU implementation
 */
#include <dgl/array.h>

#include <numeric>
#include <unordered_set>
#include <vector>

#include "../../runtime/cuda/cuda_common.h"
#include "./atomic.cuh"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;
using namespace cuda;

namespace aten {
namespace impl {

template <typename IdType>
__device__ void _warpReduce(volatile IdType* sdata, IdType tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

template <typename IdType>
__global__ void _COOGetRowNNZKernel(
    const IdType* __restrict__ row_indices, IdType* __restrict__ glb_cnt,
    const int64_t row_query, IdType nnz) {
  __shared__ IdType local_cnt[1024];
  IdType tx = threadIdx.x;
  IdType bx = blockIdx.x;
  local_cnt[tx] = 0;
  IdType start = bx * blockDim.x;
  while (start < nnz) {
    if (start + tx < nnz)
      local_cnt[tx] = (row_indices[start + tx] == row_query);
    __syncthreads();
    if (tx < 512) {
      local_cnt[tx] += local_cnt[tx + 512];
      __syncthreads();
    }
    if (tx < 256) {
      local_cnt[tx] += local_cnt[tx + 256];
      __syncthreads();
    }
    if (tx < 128) {
      local_cnt[tx] += local_cnt[tx + 128];
      __syncthreads();
    }
    if (tx < 64) {
      local_cnt[tx] += local_cnt[tx + 64];
      __syncthreads();
    }
    if (tx < 32) {
      _warpReduce(local_cnt, tx);
    }
    if (tx == 0) {
      cuda::AtomicAdd(glb_cnt, local_cnt[tx]);
    }
    start += blockDim.x * gridDim.x;
  }
}

template <DGLDeviceType XPU, typename IdType>
int64_t COOGetRowNNZ(COOMatrix coo, int64_t row) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const auto& ctx = coo.row->ctx;
  IdType nnz = coo.row->shape[0];
  IdType nt = 1024;
  IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
  NDArray rst = NDArray::Empty({1}, coo.row->dtype, coo.row->ctx);
  _Fill(rst.Ptr<IdType>(), 1, IdType(0));
  CUDA_KERNEL_CALL(
      _COOGetRowNNZKernel, nb, nt, 0, stream, coo.row.Ptr<IdType>(),
      rst.Ptr<IdType>(), row, nnz);
  rst = rst.CopyTo(DGLContext{kDGLCPU, 0});
  return *rst.Ptr<IdType>();
}

template int64_t COOGetRowNNZ<kDGLCUDA, int32_t>(COOMatrix, int64_t);
template int64_t COOGetRowNNZ<kDGLCUDA, int64_t>(COOMatrix, int64_t);

template <typename IdType>
__global__ void _COOGetAllRowNNZKernel(
    const IdType* __restrict__ row_indices, IdType* __restrict__ glb_cnts,
    IdType nnz) {
  IdType eid = blockIdx.x * blockDim.x + threadIdx.x;
  while (eid < nnz) {
    IdType row = row_indices[eid];
    cuda::AtomicAdd(glb_cnts + row, IdType(1));
    eid += blockDim.x * gridDim.x;
  }
}

template <DGLDeviceType XPU, typename IdType>
NDArray COOGetRowNNZ(COOMatrix coo, NDArray rows) {
  hipStream_t stream = runtime::getCurrentCUDAStream();
  const auto& ctx = coo.row->ctx;
  IdType nnz = coo.row->shape[0];
  IdType num_rows = coo.num_rows;
  IdType num_queries = rows->shape[0];
  if (num_queries == 1) {
    auto rows_cpu = rows.CopyTo(DGLContext{kDGLCPU, 0});
    int64_t row = *rows_cpu.Ptr<IdType>();
    IdType nt = 1024;
    IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
    NDArray rst = NDArray::Empty({1}, coo.row->dtype, coo.row->ctx);
    _Fill(rst.Ptr<IdType>(), 1, IdType(0));
    CUDA_KERNEL_CALL(
        _COOGetRowNNZKernel, nb, nt, 0, stream, coo.row.Ptr<IdType>(),
        rst.Ptr<IdType>(), row, nnz);
    return rst;
  } else {
    IdType nt = 1024;
    IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
    NDArray in_degrees = NDArray::Empty({num_rows}, rows->dtype, rows->ctx);
    _Fill(in_degrees.Ptr<IdType>(), num_rows, IdType(0));
    CUDA_KERNEL_CALL(
        _COOGetAllRowNNZKernel, nb, nt, 0, stream, coo.row.Ptr<IdType>(),
        in_degrees.Ptr<IdType>(), nnz);
    return IndexSelect(in_degrees, rows);
  }
}

template NDArray COOGetRowNNZ<kDGLCUDA, int32_t>(COOMatrix, NDArray);
template NDArray COOGetRowNNZ<kDGLCUDA, int64_t>(COOMatrix, NDArray);

}  // namespace impl
}  // namespace aten
}  // namespace dgl
