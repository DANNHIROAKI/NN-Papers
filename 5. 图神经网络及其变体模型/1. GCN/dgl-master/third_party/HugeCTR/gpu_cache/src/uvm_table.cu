#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>
#include <immintrin.h>

#include <atomic>
#include <iostream>
#include <limits>
#include <mutex>
#include <uvm_table.hpp>

namespace cg = cooperative_groups;

namespace {

constexpr int set_size = 4;
constexpr int block_size = 256;

template <typename key_type>
__host__ __device__ key_type hash(key_type key) {
  return key;
}

template <typename key_type>
__global__ void hash_add_kernel(const key_type* new_keys, const int num_keys, key_type* keys,
                                const int num_sets, int* set_sizes, const int max_set_size,
                                key_type* missing_keys, int* num_missing_keys) {
  __shared__ key_type s_missing_keys[block_size];
  __shared__ int s_missing_count;
  __shared__ size_t s_missing_idx;

  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();

  if (block.thread_rank() == 0) {
    s_missing_count = 0;
  }
  block.sync();

  size_t idx = grid.thread_rank();
  if (idx < num_keys) {
    auto key = new_keys[idx];
    size_t idx_set = hash(key) % num_sets;
    int prev_set_size = atomicAdd(&set_sizes[idx_set], 1);
    if (prev_set_size < max_set_size) {
      keys[idx_set * max_set_size + prev_set_size] = key;
    } else {
      int count = atomicAdd(&s_missing_count, 1);
      s_missing_keys[count] = key;
    }
  }

  block.sync();
  if (block.thread_rank() == 0) {
    s_missing_idx = atomicAdd(num_missing_keys, s_missing_count);
  }
  block.sync();
  for (size_t i = block.thread_rank(); i < s_missing_count; i += block.num_threads()) {
    missing_keys[s_missing_idx + i] = s_missing_keys[i];
  }
}

template <typename key_type, typename index_type>
__global__ void hash_query_kernel(const key_type* query_keys, int* num_keys_ptr,
                                  const key_type* keys, const size_t num_sets,
                                  const int max_set_size, index_type* output_indices) {
  constexpr int tile_size = set_size;
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto tile = cg::tiled_partition<tile_size>(block);
  int num_keys = *num_keys_ptr;
  if (num_keys == 0) return;

#if (CUDA_VERSION < 11060)
  size_t num_threads_per_grid = grid.size();
#else
  size_t num_threads_per_grid = grid.num_threads();
#endif

  size_t step = (num_keys - 1) / num_threads_per_grid + 1;
  for (size_t i = 0; i < step; i++) {
    size_t idx = i * num_threads_per_grid + grid.thread_rank();
    key_type query_key = std::numeric_limits<key_type>::max();
    if (idx < num_keys) {
      query_key = query_keys[idx];
    }
    auto idx_set = hash(query_key) % num_sets;
    for (int j = 0; j < tile_size; j++) {
      auto current_idx_set = tile.shfl(idx_set, j);
      auto current_query_key = tile.shfl(query_key, j);
      if (current_query_key == std::numeric_limits<key_type>::max()) {
        continue;
      }
      auto candidate_key = keys[current_idx_set * set_size + tile.thread_rank()];
      int existed = tile.ballot(current_query_key == candidate_key);
      auto current_idx = tile.shfl(idx, 0) + j;
      if (existed) {
        int src_lane = __ffs(existed) - 1;
        size_t found_idx = current_idx_set * set_size + src_lane;
        output_indices[current_idx] = num_sets * src_lane + current_idx_set;
      } else {
        output_indices[current_idx] = std::numeric_limits<index_type>::max();
      }
    }
  }
}

template <typename key_type, typename index_type>
__global__ void hash_query_kernel(const key_type* query_keys, const int num_keys,
                                  const key_type* keys, const size_t num_sets,
                                  const int max_set_size, index_type* output_indices,
                                  key_type* missing_keys, int* missing_positions,
                                  int* missing_count) {
  __shared__ key_type s_missing_keys[block_size];
  __shared__ key_type s_missing_positions[block_size];
  __shared__ int s_missing_count;
  __shared__ int s_missing_idx;

  constexpr int tile_size = set_size;

  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto tile = cg::tiled_partition<tile_size>(block);

  if (block.thread_rank() == 0) {
    s_missing_count = 0;
  }
  block.sync();

  size_t idx = grid.thread_rank();
  key_type query_key = std::numeric_limits<key_type>::max();
  if (idx < num_keys) {
    query_key = query_keys[idx];
  }
  auto idx_set = hash(query_key) % num_sets;

  for (int j = 0; j < tile_size; j++) {
    auto current_idx_set = tile.shfl(idx_set, j);
    auto current_query_key = tile.shfl(query_key, j);
    if (current_query_key == std::numeric_limits<key_type>::max()) {
      continue;
    }
    auto candidate_key = keys[current_idx_set * set_size + tile.thread_rank()];
    int existed = tile.ballot(current_query_key == candidate_key);
    if (existed) {
      int src_lane = __ffs(existed) - 1;
      size_t found_idx = current_idx_set * set_size + src_lane;
      output_indices[tile.shfl(idx, 0) + j] = num_sets * src_lane + current_idx_set;
    } else {
      auto current_idx = tile.shfl(idx, 0) + j;
      output_indices[current_idx] = std::numeric_limits<index_type>::max();
      if (tile.thread_rank() == 0) {
        int s_count = atomicAdd(&s_missing_count, 1);
        s_missing_keys[s_count] = current_query_key;
        s_missing_positions[s_count] = current_idx;
      }
    }
  }

  if (missing_keys == nullptr) {
    if (grid.thread_rank() == 0 && missing_count) {
      *missing_count = 0;
    }
    return;
  }
  block.sync();
  if (block.thread_rank() == 0) {
    s_missing_idx = atomicAdd(missing_count, s_missing_count);
  }
  block.sync();
  for (size_t i = block.thread_rank(); i < s_missing_count; i += block.num_threads()) {
    missing_keys[s_missing_idx + i] = s_missing_keys[i];
    missing_positions[s_missing_idx + i] = s_missing_positions[i];
  }
}

template <int warp_size>
__forceinline__ __device__ void warp_tile_copy(const size_t lane_idx,
                                               const size_t emb_vec_size_in_float,
                                               volatile float* d_dst, const float* d_src) {
  // 16 bytes align
  if (emb_vec_size_in_float % 4 != 0 || (size_t)d_dst % 16 != 0 || (size_t)d_src % 16 != 0) {
#pragma unroll
    for (size_t i = lane_idx; i < emb_vec_size_in_float; i += warp_size) {
      d_dst[i] = d_src[i];
    }
  } else {
#pragma unroll
    for (size_t i = lane_idx; i < emb_vec_size_in_float / 4; i += warp_size) {
      *(float4*)(d_dst + i * 4) = __ldg((const float4*)(d_src + i * 4));
    }
  }
}

template <typename index_type, typename vec_type>
__global__ void read_vectors_kernel(const index_type* query_indices, const int num_keys,
                                    const vec_type* vectors, const int vec_size,
                                    vec_type* output_vectors) {
  constexpr int warp_size = 32;

  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto tile = cg::tiled_partition<warp_size>(block);

#if (CUDA_VERSION < 11060)
  auto num_threads_per_grid = grid.size();
#else
  auto num_threads_per_grid = grid.num_threads();
#endif

  for (int step = 0; step < (num_keys - 1) / num_threads_per_grid + 1; step++) {
    int key_num = step * num_threads_per_grid + grid.thread_rank();
    index_type idx = std::numeric_limits<index_type>::max();
    if (key_num < num_keys) {
      idx = query_indices[key_num];
    }
#pragma unroll 4
    for (size_t j = 0; j < warp_size; j++) {
      index_type current_idx = tile.shfl(idx, j);
      index_type idx_write = tile.shfl(key_num, 0) + j;
      if (current_idx == std::numeric_limits<index_type>::max()) continue;
      warp_tile_copy<warp_size>(tile.thread_rank(), vec_size, output_vectors + idx_write * vec_size,
                                vectors + current_idx * vec_size);
    }
  }
}

template <typename index_type, typename vec_type>
__global__ void distribute_vectors_kernel(const index_type* postions, const size_t num_keys,
                                          const vec_type* vectors, const int vec_size,
                                          vec_type* output_vectors) {
  constexpr int warp_size = 32;

  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto tile = cg::tiled_partition<warp_size>(block);

#if (CUDA_VERSION < 11060)
  auto num_threads_per_grid = grid.size();
#else
  auto num_threads_per_grid = grid.num_threads();
#endif

  for (size_t step = 0; step < (num_keys - 1) / num_threads_per_grid + 1; step++) {
    size_t key_num = step * num_threads_per_grid + grid.thread_rank();
    index_type idx = std::numeric_limits<index_type>::max();
    if (key_num < num_keys) {
      idx = postions[key_num];
    }
#pragma unroll 4
    for (size_t j = 0; j < warp_size; j++) {
      size_t idx_write = tile.shfl(idx, j);
      size_t idx_read = tile.shfl(key_num, 0) + j;
      if (idx_write == std::numeric_limits<index_type>::max()) continue;
      warp_tile_copy<warp_size>(tile.thread_rank(), vec_size,
                                output_vectors + (size_t)idx_write * vec_size,
                                vectors + (size_t)idx_read * vec_size);
    }
  }
}

}  // namespace

namespace gpu_cache {
template <typename key_type, typename index_type, typename vec_type>
UvmTable<key_type, index_type, vec_type>::UvmTable(const size_t device_table_capacity,
                                                   const size_t host_table_capacity,
                                                   const int max_batch_size, const int vec_size,
                                                   const vec_type default_value)
    : max_batch_size_(std::max(100000, max_batch_size)),
      vec_size_(vec_size),
      num_set_((device_table_capacity - 1) / set_size + 1),
      num_host_set_((host_table_capacity - 1) / set_size + 1),
      table_capacity_(num_set_ * set_size),
      default_vector_(vec_size, default_value),
      device_table_(device_table_capacity, set_size, max_batch_size_),
      host_table_(host_table_capacity * 1.3, set_size, max_batch_size_) {
  CUDA_CHECK(hipMalloc(&d_keys_buffer_, sizeof(key_type) * max_batch_size_));
  CUDA_CHECK(hipMalloc(&d_vectors_buffer_, sizeof(vec_type) * max_batch_size_ * vec_size_));
  CUDA_CHECK(hipMalloc(&d_vectors_, sizeof(vec_type) * device_table_.capacity * vec_size_));

  CUDA_CHECK(hipMalloc(&d_output_indices_, sizeof(index_type) * max_batch_size_));
  CUDA_CHECK(hipMalloc(&d_output_host_indices_, sizeof(index_type) * max_batch_size_));
  CUDA_CHECK(hipHostMalloc(&h_output_host_indices_, sizeof(index_type) * max_batch_size_));
  CUDA_CHECK(hipMalloc(&d_missing_keys_, sizeof(key_type) * max_batch_size_));
  CUDA_CHECK(hipMalloc(&d_missing_positions_, sizeof(int) * max_batch_size_));
  CUDA_CHECK(hipMalloc(&d_missing_count_, sizeof(int)));
  CUDA_CHECK(hipMemset(d_missing_count_, 0, sizeof(int)));
  CUDA_CHECK(hipStreamCreate(&query_stream_));
  for (int i = 0; i < num_buffers_; i++) {
    int batch_size_per_buffer = ceil(1.0 * max_batch_size_ / num_buffers_);
    CUDA_CHECK(
        hipHostMalloc(&h_cpy_buffers_[i], sizeof(vec_type) * batch_size_per_buffer * vec_size));
    CUDA_CHECK(hipMalloc(&d_cpy_buffers_[i], sizeof(vec_type) * batch_size_per_buffer * vec_size));
    CUDA_CHECK(hipStreamCreate(&cpy_streams_[i]));
    CUDA_CHECK(hipEventCreate(&cpy_events_[i]));
  }
  CUDA_CHECK(hipHostMalloc(&h_missing_keys_, sizeof(key_type) * max_batch_size_));
  CUDA_CHECK(hipEventCreate(&query_event_));
  h_vectors_.resize(host_table_.capacity * vec_size_);
}

template <typename key_type, typename index_type, typename vec_type>
void UvmTable<key_type, index_type, vec_type>::add(const key_type* h_keys,
                                                   const vec_type* h_vectors,
                                                   const size_t num_keys) {
  std::vector<key_type> h_missing_keys;
  size_t num_batches = (num_keys - 1) / max_batch_size_ + 1;
  for (size_t i = 0; i < num_batches; i++) {
    size_t this_batch_size =
        i != num_batches - 1 ? max_batch_size_ : num_keys - i * max_batch_size_;
    CUDA_CHECK(hipMemcpy(d_keys_buffer_, h_keys + i * max_batch_size_,
                          sizeof(*d_keys_buffer_) * this_batch_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_missing_count_, 0, sizeof(*d_missing_count_)));
    device_table_.add(d_keys_buffer_, this_batch_size, d_missing_keys_, d_missing_count_, 0);
    CUDA_CHECK(hipDeviceSynchronize());
    int num_missing_keys;
    CUDA_CHECK(hipMemcpy(&num_missing_keys, d_missing_count_, sizeof(num_missing_keys),
                          hipMemcpyDeviceToHost));
    size_t prev_size = h_missing_keys.size();
    h_missing_keys.resize(prev_size + num_missing_keys);
    CUDA_CHECK(hipMemcpy(h_missing_keys.data() + prev_size, d_missing_keys_,
                          sizeof(*d_missing_keys_) * num_missing_keys, hipMemcpyDeviceToHost));
  }

  std::vector<key_type> h_final_missing_keys;
  num_batches = h_missing_keys.size() ? (h_missing_keys.size() - 1) / max_batch_size_ + 1 : 0;
  for (size_t i = 0; i < num_batches; i++) {
    size_t this_batch_size =
        i != num_batches - 1 ? max_batch_size_ : h_missing_keys.size() - i * max_batch_size_;
    CUDA_CHECK(hipMemcpy(d_keys_buffer_, h_missing_keys.data() + i * max_batch_size_,
                          sizeof(*d_keys_buffer_) * this_batch_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_missing_count_, 0, sizeof(*d_missing_count_)));
    host_table_.add(d_keys_buffer_, this_batch_size, d_missing_keys_, d_missing_count_, 0);
    CUDA_CHECK(hipDeviceSynchronize());
    int num_missing_keys;
    CUDA_CHECK(hipMemcpy(&num_missing_keys, d_missing_count_, sizeof(num_missing_keys),
                          hipMemcpyDeviceToHost));
    size_t prev_size = h_final_missing_keys.size();
    h_final_missing_keys.resize(prev_size + num_missing_keys);
    CUDA_CHECK(hipMemcpy(h_final_missing_keys.data() + prev_size, d_missing_keys_,
                          sizeof(*d_missing_keys_) * num_missing_keys, hipMemcpyDeviceToHost));
  }

  std::vector<key_type> h_keys_buffer(max_batch_size_);
  std::vector<index_type> h_indices_buffer(max_batch_size_);
  std::vector<int> h_positions_buffer(max_batch_size_);

  num_batches = (num_keys - 1) / max_batch_size_ + 1;

  size_t num_hit_keys = 0;
  for (size_t i = 0; i < num_batches; i++) {
    size_t this_batch_size =
        i != num_batches - 1 ? max_batch_size_ : num_keys - i * max_batch_size_;
    CUDA_CHECK(hipMemcpy(d_keys_buffer_, h_keys + i * max_batch_size_,
                          sizeof(*d_keys_buffer_) * this_batch_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_missing_count_, 0, sizeof(*d_missing_count_)));
    device_table_.query(d_keys_buffer_, this_batch_size, d_output_indices_, d_missing_keys_,
                        d_missing_positions_, d_missing_count_, 0);
    CUDA_CHECK(hipStreamSynchronize(0));

    CUDA_CHECK(hipMemcpy(d_vectors_buffer_, h_vectors + i * max_batch_size_ * vec_size_,
                          sizeof(*d_vectors_) * this_batch_size * vec_size_,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipStreamSynchronize(0));
    if (num_hit_keys < device_table_.capacity) {
      distribute_vectors_kernel<<<(this_batch_size - 1) / block_size + 1, block_size, 0, 0>>>(
          d_output_indices_, this_batch_size, d_vectors_buffer_, vec_size_, d_vectors_);
      CUDA_CHECK(hipStreamSynchronize(0));
    }

    int num_missing_keys;
    CUDA_CHECK(hipMemcpy(&num_missing_keys, d_missing_count_, sizeof(num_missing_keys),
                          hipMemcpyDeviceToHost));
    num_hit_keys += this_batch_size - num_missing_keys;
    host_table_.query(d_missing_keys_, num_missing_keys, d_output_indices_, nullptr, nullptr,
                      nullptr, 0);

    CUDA_CHECK(hipMemcpy(h_keys_buffer.data(), d_missing_keys_,
                          sizeof(*d_missing_keys_) * num_missing_keys, hipMemcpyDeviceToHost))

    CUDA_CHECK(hipMemcpy(h_indices_buffer.data(), d_output_indices_,
                          sizeof(*d_output_indices_) * num_missing_keys, hipMemcpyDeviceToHost))

    CUDA_CHECK(hipMemcpy(h_positions_buffer.data(), d_missing_positions_,
                          sizeof(*d_missing_positions_) * num_missing_keys, hipMemcpyDeviceToHost))

    for (int j = 0; j < num_missing_keys; j++) {
      if (h_indices_buffer[j] != std::numeric_limits<index_type>::max()) {
        memcpy(h_vectors_.data() + h_indices_buffer[j] * vec_size_,
               h_vectors + (i * max_batch_size_ + h_positions_buffer[j]) * vec_size_,
               sizeof(*h_vectors) * vec_size_);
      } else {
        size_t prev_idx = h_vectors_.size() / vec_size_;
        h_final_missing_items_.emplace(h_keys_buffer[j], prev_idx);
        h_vectors_.resize(h_vectors_.size() + vec_size_);
        memcpy(h_vectors_.data() + prev_idx * vec_size_,
               h_vectors + (i * max_batch_size_ + h_positions_buffer[j]) * vec_size_,
               sizeof(*h_vectors) * vec_size_);
      }
    }
  }
  CUDA_CHECK(hipMemset(d_missing_count_, 0, sizeof(*d_missing_count_)));
}

template <typename key_type, typename index_type, typename vec_type>
void UvmTable<key_type, index_type, vec_type>::query(const key_type* d_keys, const int num_keys,
                                                     vec_type* d_vectors, hipStream_t stream) {
  if (!num_keys) return;
  CUDA_CHECK(hipEventRecord(query_event_, stream));
  CUDA_CHECK(hipStreamWaitEvent(query_stream_, query_event_));

  static_assert(num_buffers_ >= 2);
  device_table_.query(d_keys, num_keys, d_output_indices_, d_missing_keys_, d_missing_positions_,
                      d_missing_count_, query_stream_);

  CUDA_CHECK(hipEventRecord(query_event_, query_stream_));
  CUDA_CHECK(hipStreamWaitEvent(cpy_streams_[0], query_event_));

  int num_missing_keys;
  CUDA_CHECK(hipMemcpyAsync(&num_missing_keys, d_missing_count_, sizeof(*d_missing_count_),
                             hipMemcpyDeviceToHost, cpy_streams_[0]));

  host_table_.query(d_missing_keys_, d_missing_count_, d_output_host_indices_, query_stream_);
  CUDA_CHECK(hipStreamSynchronize(cpy_streams_[0]));

  CUDA_CHECK(hipMemsetAsync(d_missing_count_, 0, sizeof(*d_missing_count_), query_stream_));

  CUDA_CHECK(hipMemcpyAsync(h_output_host_indices_, d_output_host_indices_,
                             sizeof(index_type) * num_missing_keys, hipMemcpyDeviceToHost,
                             query_stream_));

  CUDA_CHECK(hipMemcpyAsync(h_missing_keys_, d_missing_keys_, sizeof(key_type) * num_missing_keys,
                             hipMemcpyDeviceToHost, cpy_streams_[0]));

  read_vectors_kernel<<<(num_keys - 1) / block_size + 1, block_size, 0, cpy_streams_[1]>>>(
      d_output_indices_, num_keys, d_vectors_, vec_size_, d_vectors);

  CUDA_CHECK(hipStreamSynchronize(query_stream_));
  CUDA_CHECK(hipStreamSynchronize(cpy_streams_[0]));

  int num_keys_per_buffer = ceil(1.0 * num_missing_keys / num_buffers_);

  for (int buffer_num = 0; buffer_num < num_buffers_; buffer_num++) {
    int num_keys_this_buffer = buffer_num != num_buffers_ - 1
                                   ? num_keys_per_buffer
                                   : num_missing_keys - num_keys_per_buffer * buffer_num;
    if (!num_keys_this_buffer) break;
#pragma omp parallel for num_threads(8)
    for (size_t i = 0; i < static_cast<size_t>(num_keys_this_buffer); i++) {
      size_t idx_key = buffer_num * num_keys_per_buffer + i;
      index_type index = h_output_host_indices_[idx_key];
      if (index == std::numeric_limits<index_type>::max()) {
        key_type key = h_missing_keys_[idx_key];
        auto iterator = h_final_missing_items_.find(key);
        if (iterator != h_final_missing_items_.end()) {
          index = iterator->second;
        }
      }
      if (index != std::numeric_limits<index_type>::max()) {
        memcpy(h_cpy_buffers_[buffer_num] + i * vec_size_, h_vectors_.data() + index * vec_size_,
               sizeof(vec_type) * vec_size_);
      } else {
        memcpy(h_cpy_buffers_[buffer_num] + i * vec_size_, default_vector_.data(),
               sizeof(vec_type) * vec_size_);
      }
    }
    CUDA_CHECK(hipMemcpyAsync(d_cpy_buffers_[buffer_num], h_cpy_buffers_[buffer_num],
                               sizeof(vec_type) * num_keys_this_buffer * vec_size_,
                               hipMemcpyHostToDevice, cpy_streams_[buffer_num]));

    distribute_vectors_kernel<<<(num_keys_this_buffer - 1) / block_size + 1, block_size, 0,
                                cpy_streams_[buffer_num]>>>(
        d_missing_positions_ + buffer_num * num_keys_per_buffer, num_keys_this_buffer,
        d_cpy_buffers_[buffer_num], vec_size_, d_vectors);
  }

  for (int i = 0; i < num_buffers_; i++) {
    CUDA_CHECK(hipEventRecord(cpy_events_[i], cpy_streams_[i]));
    CUDA_CHECK(hipStreamWaitEvent(stream, cpy_events_[i]));
  }
}

template <typename key_type, typename index_type, typename vec_type>
void UvmTable<key_type, index_type, vec_type>::clear(hipStream_t stream) {
  device_table_.clear(stream);
  host_table_.clear(stream);
}

template <typename key_type, typename index_type, typename vec_type>
UvmTable<key_type, index_type, vec_type>::~UvmTable() {
  CUDA_CHECK(hipFree(d_keys_buffer_));
  CUDA_CHECK(hipFree(d_vectors_buffer_));
  CUDA_CHECK(hipFree(d_vectors_));

  CUDA_CHECK(hipFree(d_output_indices_));
  CUDA_CHECK(hipFree(d_output_host_indices_));
  CUDA_CHECK(hipHostFree(h_output_host_indices_));

  CUDA_CHECK(hipFree(d_missing_keys_));
  CUDA_CHECK(hipFree(d_missing_positions_));
  CUDA_CHECK(hipFree(d_missing_count_));
  CUDA_CHECK(hipHostFree(h_missing_keys_));

  CUDA_CHECK(hipStreamDestroy(query_stream_));
  CUDA_CHECK(hipEventDestroy(query_event_));

  for (int i = 0; i < num_buffers_; i++) {
    CUDA_CHECK(hipHostFree(h_cpy_buffers_[i]));
    CUDA_CHECK(hipFree(d_cpy_buffers_[i]));
    CUDA_CHECK(hipStreamDestroy(cpy_streams_[i]));
    CUDA_CHECK(hipEventDestroy(cpy_events_[i]));
  }
}

template <typename key_type, typename index_type>
HashBlock<key_type, index_type>::HashBlock(size_t expected_capacity, int set_size, int batch_size)
    : max_set_size_(set_size), batch_size_(batch_size) {
  if (expected_capacity) {
    num_sets = (expected_capacity - 1) / set_size + 1;
  } else {
    num_sets = 10000;
  }
  capacity = num_sets * set_size;
  CUDA_CHECK(hipMalloc(&keys, sizeof(*keys) * capacity));
  CUDA_CHECK(hipMalloc(&set_sizes_, sizeof(*set_sizes_) * num_sets));
  CUDA_CHECK(hipMemset(set_sizes_, 0, sizeof(*set_sizes_) * num_sets));
}

template <typename key_type, typename index_type>
HashBlock<key_type, index_type>::~HashBlock() {
  CUDA_CHECK(hipFree(keys));
  CUDA_CHECK(hipFree(set_sizes_));
}

template <typename key_type, typename index_type>
void HashBlock<key_type, index_type>::query(const key_type* query_keys, const size_t num_keys,
                                            index_type* output_indices, key_type* missing_keys,
                                            int* missing_positions, int* num_missing_keys,
                                            hipStream_t stream) {
  if (num_keys == 0) {
    return;
  }
  size_t num_batches = (num_keys - 1) / batch_size_ + 1;
  for (size_t i = 0; i < num_batches; i++) {
    size_t this_batch_size = i != num_batches - 1 ? batch_size_ : num_keys - i * batch_size_;
    hash_query_kernel<<<(this_batch_size - 1) / block_size + 1, block_size, 0, stream>>>(
        query_keys, this_batch_size, keys, num_sets, max_set_size_, output_indices, missing_keys,
        missing_positions, num_missing_keys);
  }
}

template <typename key_type, typename index_type>
void HashBlock<key_type, index_type>::query(const key_type* query_keys, int* num_keys,
                                            index_type* output_indices, hipStream_t stream) {
  hash_query_kernel<<<128, 64, 0, stream>>>(query_keys, num_keys, keys, num_sets, max_set_size_,
                                            output_indices);
}

template <typename key_type, typename index_type>
void HashBlock<key_type, index_type>::add(const key_type* new_keys, const size_t num_keys,
                                          key_type* missing_keys, int* num_missing_keys,
                                          hipStream_t stream) {
  if (num_keys == 0) {
    return;
  }
  size_t num_batches = (num_keys - 1) / batch_size_ + 1;
  for (size_t i = 0; i < num_batches; i++) {
    size_t this_batch_size = i != num_batches - 1 ? batch_size_ : num_keys - i * batch_size_;
    hash_add_kernel<<<(this_batch_size - 1) / block_size + 1, block_size, 0, stream>>>(
        new_keys + i * this_batch_size, this_batch_size, keys, num_sets, set_sizes_, max_set_size_,
        missing_keys, num_missing_keys);
  }
}

template <typename key_type, typename index_type>
void HashBlock<key_type, index_type>::clear(hipStream_t stream) {
  CUDA_CHECK(hipMemsetAsync(set_sizes_, 0, sizeof(*set_sizes_) * num_sets, stream));
}

template class HashBlock<int, size_t>;
template class HashBlock<int64_t, size_t>;
template class HashBlock<size_t, size_t>;
template class HashBlock<unsigned int, size_t>;
template class HashBlock<long long, size_t>;

template class UvmTable<int, size_t>;
template class UvmTable<int64_t, size_t>;
template class UvmTable<size_t, size_t>;
template class UvmTable<unsigned int, size_t>;
template class UvmTable<long long, size_t>;
}  // namespace gpu_cache