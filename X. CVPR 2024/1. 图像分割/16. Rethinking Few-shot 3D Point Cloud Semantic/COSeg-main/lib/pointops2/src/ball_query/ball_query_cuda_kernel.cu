#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "ball_query_cuda_kernel.h"


__device__ int get_bt_idx_in_ball_query(int idx, const int *offset)
{
    int i = 0;
    while (1)
    {
        if (idx < offset[i])
            break;
        else
            i++;
    }
    return i;
}


__global__ void ball_query_cuda_kernel(int m, float radius, int nsample, const float *__restrict__ xyz, const float *__restrict__ new_xyz, const int *__restrict__ offset, const int *__restrict__ new_offset, int *__restrict__ idx, float *__restrict__ dist2) {
    // input: xyz (n, 3) new_xyz (m, 3)
    // output: idx (m, nsample) dist2 (m, nsample)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= m) return;

    new_xyz += pt_idx * 3;
    idx += pt_idx * nsample;
    dist2 += pt_idx * nsample;
    int bt_idx = get_bt_idx_in_ball_query(pt_idx, new_offset);
    int start;
    if (bt_idx == 0)
        start = 0;
    else
        start = offset[bt_idx - 1];
    int end = offset[bt_idx];

    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int count = 0;
    float radius2 = radius * radius;

    for(int i = 0; i < nsample; i++){
        idx[i] = -1;
        dist2[i] = 1e10;
    }
    for(int i = start; i < end; i++){
        float x = xyz[i * 3 + 0];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        if (d2 <= radius2){
            idx[count] = i;
            dist2[count] = d2;
            count++;
        }
        if (count >= nsample){
            break;
        }
    }
}


void ball_query_cuda_launcher(int m, float radius, int nsample, const float *xyz, const float *new_xyz, const int *offset, const int *new_offset, int *idx, float *dist2) {
    // input: new_xyz: (m, 3), xyz: (n, 3), idx: (m, nsample)
    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    ball_query_cuda_kernel<<<blocks, threads, 0>>>(m, radius, nsample, xyz, new_xyz, offset, new_offset, idx, dist2);
}
